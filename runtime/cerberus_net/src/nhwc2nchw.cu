// kernel to convert from OpenCV channel representation to channel-first
// see: https://docs.opencv.org/2.4/doc/tutorials/core/how_to_scan_images/how_to_scan_images.html#how-the-image-matrix-is-stored-in-the-memory

const int BLOCK_SIZE = 1024;
#include <hip/hip_runtime.h>
#include <array>

__global__ void nhwc2nchwKernel(const unsigned char* __restrict__ source, float* __restrict__ dest,
    int channelSize, int channelsNum, int rowElems, int rowSize)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = idx / channelsNum;
    int channel = idx % channelsNum;

    // what would the row be if we didn't have any padding
    int row = idx / rowElems;
    int col = idx % rowElems;

    // actual element - skip padding
    int sourceIdx = row * rowSize + col;
    dest[channelSize * channel + offset] = (float) source[sourceIdx] / 255.f;
}

// we expect all memory to already reside on device so no need to allocate anything
void nhwc2nchw(const unsigned char * source, float * dest, int channelSize,
    int channelsNum, int rowElems, int rowSize, hipStream_t Stream)
{
    const int nBlocks = (channelSize * channelsNum) / BLOCK_SIZE;
    nhwc2nchwKernel<<<nBlocks, BLOCK_SIZE, 0, Stream>>>(
        source, dest, channelSize, channelsNum, rowElems, rowSize);
}

template<typename scalar_t>
__global__ void normalizeChannelKernel(scalar_t* __restrict__ source,
    size_t channel_stride, scalar_t mean, scalar_t std)
{
    const int offset = threadIdx.x + blockIdx.x * blockDim.x;
    if (offset < channel_stride) { source[offset] = (source[offset] - mean) / std; }
}

template<typename scalar_t, size_t n_ch>
void normalize_image_chw(scalar_t* image, size_t ch_stride, const std::array<scalar_t, n_ch> &mean,
    const std::array<scalar_t, n_ch> &std, hipStream_t Stream)
{
    const int nBlocks = ch_stride / BLOCK_SIZE;
    for (size_t ch=0; ch < n_ch; ++ch)
    {
        normalizeChannelKernel<scalar_t><<<nBlocks, BLOCK_SIZE, 0, Stream>>>(
            &image[ch*ch_stride], ch_stride, mean[ch], std[ch]);
    }
}

template void normalize_image_chw<float, 3ul>(float*, size_t, std::array<float, 3ul> const&,
    std::array<float, 3ul> const&, hipStream_t);

template<typename scalar_t, typename intergral_t>
__global__ void argmax_chw_Kernel(const scalar_t* __restrict__ source,
    intergral_t* __restrict__ output, const size_t channel_stride, const size_t n_classes)
{
    const int offset = threadIdx.x + blockIdx.x * blockDim.x;
    scalar_t best_score = 0;
    intergral_t best_cls = n_classes+1;
    for (size_t cls=0; cls<n_classes; ++cls)
    {
        if (source[offset + cls*channel_stride] > best_score)
        {
            best_score = source[offset + cls*channel_stride];
            best_cls = cls;
        }
    }
    output[offset] = best_cls;
}

template<typename scalar_t, typename intergral_t>
void argmax_chw(const scalar_t* input, intergral_t* output,
    size_t n_classes, size_t ch_stride, hipStream_t Stream)
{
    const int nBlocks = ch_stride / BLOCK_SIZE;
    argmax_chw_Kernel<scalar_t, intergral_t><<<nBlocks, BLOCK_SIZE, 0, Stream>>>(
        input, output, ch_stride, n_classes);
}

template void argmax_chw<float, unsigned char>(
    const float*, unsigned char*, size_t, size_t, hipStream_t);