#include "hip/hip_runtime.h"
#include "scatter_nd.hpp"
#include "trt_utils.hpp"
#include "hip/hip_fp16.h"

#include <array>
#include <algorithm>
#include <limits>
#include <cassert>

int ScatterNDPlugin::enqueue(const nvinfer1::PluginTensorDesc* inputDesc, const nvinfer1::PluginTensorDesc* outputDesc,
	const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream)
{
    // Source code from ONNX repo implementation
    // # Compute output
    // output = np.copy(data)
    // for i in np.ndindex(indices.shape[:-1]):
    //     # NOTE: The order of iteration in this loop is not specified.
    //     # In particular, indices should not have duplicate entries: that is, if idx1 != idx2, then indices[idx1] != indices[idx2].
    //     # This ensures that the output value does not depend on the iteration order.
    //     output[indices[i]] = updates[i]
    // return output

    // Copy the entire input to the output before update
    size_t mem_size = inputDesc[0].type == nvinfer1::DataType::kFLOAT ? sizeof(float) : sizeof(__half);
    for (size_t i=0; i<inputDesc[0].dims.nbDims; i++)
    {
        mem_size *= inputDesc[0].dims.d[i];
    }
    NV_CUDA_CHECK(hipMemcpy(outputs[0], inputs[0], mem_size, hipMemcpyDeviceToDevice));

    // Checking what is actually being given as update indicies, its seems like the channel stays constant
    // and the elements are being iterated over, therefore we should just be able to copy the entire channel.
    // std::cout << "ScatterND\n";
    // for (size_t i=0; i<130; i++)
    // {
    //     std::array<int32_t, 4> data;
    //     const size_t dim_stride = 4 * sizeof(int32_t);
    //     NV_CUDA_CHECK(hipMemcpy(data.data(), inputs[1] + i * dim_stride, dim_stride, hipMemcpyDeviceToHost));
    
    //     std::for_each(data.begin(), data.end(), [](const auto& elem){ std::cout << elem << ", ";} );
    //     std::cout << "\n";
    // }
    // std::cout << std::endl;

    std::array<int32_t, 4> indicies_element;
    const size_t dim_stride = 4 * sizeof(int32_t);
    NV_CUDA_CHECK(hipMemcpy(indicies_element.data(), inputs[1], dim_stride, hipMemcpyDeviceToHost));

    const size_t elem_size = inputDesc[0].type == nvinfer1::DataType::kFLOAT ? sizeof(float) : sizeof(__half);
    const size_t channel_stride = inputDesc[0].dims.d[2] * inputDesc[0].dims.d[3] * elem_size;
    
    // indicies_element[1] contains the channel indx that is being updated.
    NV_CUDA_CHECK(hipMemcpy(
        outputs[0] + indicies_element[1] * elem_size * channel_stride, inputs[2],
        elem_size * channel_stride, hipMemcpyDeviceToDevice));

    return hipGetLastError();
}
