#include "hip/hip_runtime.h"
#include "scatter_nd.hpp"
#include "trt_utils.hpp"
#include "hip/hip_fp16.h"

template <typename scalar_t, typename intergral_t>
__global__ void ScatterND_Kernel(scalar_t* output, const intergral_t* __restrict__ indicies,
    const scalar_t* __restrict__ updates, size_t out_channels, size_t out_height, size_t out_width, size_t ind_channels)
{
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < out_height * out_width) {
        const size_t out_n_stride = out_channels * out_height * out_width;
        const size_t out_c_stride = out_height * out_width;
        const size_t out_h_stride = out_width;

        const size_t ind_batch_offset = blockIdx.y * 4 * ind_channels * out_height * out_width;
        for (size_t ch=0; ch<ind_channels; ++ch)
        {
            size_t indicies_indx = ind_batch_offset + 4 * (ch * out_c_stride + tid);
            const intergral_t output_n = indicies[indicies_indx];
            const intergral_t output_c = indicies[++indicies_indx];
            const intergral_t output_h = indicies[++indicies_indx];
            const intergral_t output_w = indicies[++indicies_indx];
            output[out_n_stride*output_n + out_c_stride*output_c + out_h_stride*output_h + output_w] = 
                updates[ch * out_c_stride + tid];
        }
    }
}

int ScatterNDPlugin::enqueue(const nvinfer1::PluginTensorDesc* inputDesc, const nvinfer1::PluginTensorDesc* outputDesc,
	const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream)
{
    // Source code from ONNX repo implementation
    // # Compute output
    // output = np.copy(data)
    // for i in np.ndindex(indices.shape[:-1]):
    //     # NOTE: The order of iteration in this loop is not specified.
    //     # In particular, indices should not have duplicate entries: that is, if idx1 != idx2, then indices[idx1] != indices[idx2].
    //     # This ensures that the output value does not depend on the iteration order.
    //     output[indices[i]] = updates[i]
    // return output

    // Copy the entire input to the output before update
    size_t mem_size = inputDesc[0].type == nvinfer1::DataType::kFLOAT ? sizeof(float) : sizeof(__half);
    for (size_t i=0; i<inputDesc[0].dims.nbDims; i++)
    {
        mem_size *= inputDesc[0].dims.d[i];
    }
    NV_CUDA_CHECK(hipMemcpyAsync(outputs[0], inputs[0], mem_size, hipMemcpyDeviceToDevice, stream));

    // std::array<int32_t, 4> indicies_element;
    // const size_t dim_stride = 4 * sizeof(int32_t);
    // NV_CUDA_CHECK(hipMemcpy(indicies_element.data(), inputs[1], dim_stride, hipMemcpyDeviceToHost));

    // const size_t elem_size = inputDesc[0].type == nvinfer1::DataType::kFLOAT ? sizeof(float) : sizeof(__half);
    // const size_t channel_stride = inputDesc[0].dims.d[2] * inputDesc[0].dims.d[3] * elem_size;
    
    // // indicies_element[1] contains the channel indx that is being updated.
    // NV_CUDA_CHECK(hipMemcpy(
    //     outputs[0] + indicies_element[1] * elem_size * channel_stride, inputs[2],
    //     elem_size * channel_stride, hipMemcpyDeviceToDevice));

    const dim3 nBlocks{ inputDesc[0].dims.d[2] * inputDesc[0].dims.d[3] / CUDA_NUM_THREADS, inputDesc[0].dims.d[0], 1 };

    switch(inputDesc[0].type)
	{
		case nvinfer1::DataType::kFLOAT:
		{
            ScatterND_Kernel<<<nBlocks, CUDA_NUM_THREADS, 0, stream>>>(
                reinterpret_cast<float*>(outputs[0]),
                reinterpret_cast<const int32_t*>(inputs[1]),
                reinterpret_cast<const float*>(inputs[2]),
                inputDesc[0].dims.d[1], inputDesc[0].dims.d[2],
                inputDesc[0].dims.d[3], inputDesc[1].dims.d[1]);
            break;
        }
		case nvinfer1::DataType::kHALF:
		{
            ScatterND_Kernel<<<nBlocks, CUDA_NUM_THREADS, 0, stream>>>(
                reinterpret_cast<__half*>(outputs[0]),
                reinterpret_cast<const int32_t*>(inputs[1]),
                reinterpret_cast<const __half*>(inputs[2]),
                inputDesc[0].dims.d[1], inputDesc[0].dims.d[2],
                inputDesc[0].dims.d[3], inputDesc[1].dims.d[1]);
            break;
        }
        default:
            throw( std::runtime_error{"ScatterNDPlugin Unsupported Input Type"} );
            break;
    }

    return hipGetLastError();
}
