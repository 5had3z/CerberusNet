#include "hip/hip_runtime.h"
#include "grid_sampler.hpp"
#include "trt_utils.hpp"
#include "hip/hip_fp16.h"

#include <limits>
#include <cassert>

// CUDA: grid stride looping
//
// int64_t _i_n_d_e_x specifically prevents overflow in the loop increment.
// If input.numel() < INT_MAX, _i_n_d_e_x < INT_MAX, except after the final
// iteration of the loop where _i_n_d_e_x += blockDim.x * gridDim.x can be
// greater than INT_MAX.  But in that case _i_n_d_e_x >= n, so there are no
// further iterations and the overflowed value in i=_i_n_d_e_x is not used.
#define CUDA_KERNEL_LOOP_TYPE(i, n, index_type)                         \
  int64_t _i_n_d_e_x = blockIdx.x * blockDim.x + threadIdx.x;           \
  for (index_type i=_i_n_d_e_x; _i_n_d_e_x < (n); _i_n_d_e_x+=blockDim.x * gridDim.x, i=_i_n_d_e_x)

#define CUDA_KERNEL_LOOP(i, n) CUDA_KERNEL_LOOP_TYPE(i, n, int)

// CUDA: number of blocks for threads.
inline int GET_BLOCKS(const int64_t N) {
  assert(N > 0 && "CUDA kernel launch blocks must be positive");
  constexpr int64_t max_int = std::numeric_limits<int>::max();

  // Round up division for positive number that cannot cause integer overflow
  auto block_num = (N - 1) / CUDA_NUM_THREADS + 1;
  assert(block_num <= max_int && "Can't schedule too many blocks on CUDA device");

  return static_cast<int>(block_num);
}

static __forceinline__ __device__
bool within_bounds_2d(int h, int w, int H, int W) {
  return h >= 0 && h < H && w >= 0 && w < W;
}

// Unnormalizes a coordinate from the -1 to +1 scale to its pixel index value,
// where we view each pixel as an area between (idx - 0.5) and (idx + 0.5).
// if align_corners: -1 and +1 get sent to the centers of the corner pixels
//     -1 --> 0
//     +1 --> (size - 1)
//     scale_factor = (size - 1) / 2
// if not align_corners: -1 and +1 get sent to the image edges
//     -1 --> -0.5
//     +1 --> (size - 1) + 0.5 == size - 0.5
//     scale_factor = size / 2
template <typename scalar_t>
static __forceinline__ __device__
scalar_t grid_sampler_unnormalize(scalar_t coord, int size, bool align_corners)
{
    if (align_corners) {
        // unnormalize coord from [-1, 1] to [0, size - 1]
        return ((coord + static_cast<scalar_t>(1.f)) / static_cast<scalar_t>(2)) * static_cast<scalar_t>(size - 1);
    } else {
        // unnormalize coord from [-1, 1] to [-0.5, size - 0.5]
        return ((coord + static_cast<scalar_t>(1.f)) * static_cast<scalar_t>(size - 1)) / static_cast<scalar_t>(2);
    }
}

// Clips coordinates to between 0 and clip_limit - 1
template <typename scalar_t>
static __forceinline__ __device__
scalar_t clip_coordinates(scalar_t in, int clip_limit) {
    return ::min(static_cast<scalar_t>(clip_limit - 1), ::max(in, static_cast<scalar_t>(0)));
}

// Reflects coordinates until they fall between low and high (inclusive).
// The bounds are passed as twice their value so that half-integer values
// can be represented as ints.
template <typename scalar_t>
static __forceinline__ __device__
scalar_t reflect_coordinates(scalar_t in, int twice_low, int twice_high)
{
    if (twice_low == twice_high) { return static_cast<scalar_t>(0); }

    scalar_t min = static_cast<scalar_t>(twice_low) / 2;
    scalar_t span = static_cast<scalar_t>(twice_high - twice_low) / 2;
    in = ::fabs(in - min);

    // `fmod` returns same sign as `in`, which is positive after the `fabs` above.
    scalar_t extra = ::fmod(in, span);
    int flips = static_cast<int>(::floor(in / span));

    return flips % 2 == 0 ? extra + min : span - extra + min;
}

template<typename scalar_t> 
static __forceinline__ __device__ 
scalar_t safe_downgrade_to_int_range(scalar_t x){
    // -100.0 does not have special meaning. This is just to make sure 
    // it's not within_bounds_2d or within_bounds_3d, and does not cause 
    // undefined behavior. See #35506.  
    if (x > INT_MAX-1 || x < INT_MIN || !::isfinite(static_cast<double>(x))) 
        return static_cast<scalar_t>(-100.0); 
    return x;
}

// Computes the pixel source index value for a grid coordinate
template <typename scalar_t>
static __forceinline__ __device__
scalar_t grid_sampler_compute_source_index(scalar_t coord, int size,
    GridSampler::Padding padding_mode, bool align_corners)
{
    coord = grid_sampler_unnormalize(coord, size, align_corners);
    if (padding_mode == GridSampler::Padding::Border) {
        // clip coordinates to image borders
        coord = clip_coordinates(coord, size);
    }
    else if (padding_mode == GridSampler::Padding::Reflection) {
        // reflect coordinates by image borders
        if (align_corners) {
            coord = reflect_coordinates(coord, 0, 2*(size - 1));
        } else {
            coord = reflect_coordinates(coord, -1, 2*size - 1);
        }
        // clip coordinates to image borders
        coord = clip_coordinates(coord, size);
    }

    coord = safe_downgrade_to_int_range(coord); 
    return coord;
}

template <typename scalar_t>
__global__ void grid_sampler_kernel(const size_t nthreads,
    const scalar_t* __restrict__ input, size_t C, size_t inp_H, size_t inp_W,
    const scalar_t* __restrict__ grid,
    scalar_t* output, size_t out_H, size_t out_W,
    const GridSampler::Interpolation interpolation_mode,
    const GridSampler::Padding padding_mode, bool align_corners)
{
    // Input Strides
    size_t inp_sN = C * inp_H * inp_W;
    size_t inp_sC = inp_H * inp_W;
    size_t inp_sH = inp_W;
    size_t inp_sW = 1;

    // Grid Strides
    size_t grid_sN = inp_H * inp_W * 2;
    size_t grid_sH = inp_W * 2;
    size_t grid_sW = 2;
    size_t grid_sCoor = 1;

    // Output Strides
    size_t out_sN = C * out_H * out_W;
    size_t out_sC = out_H * out_W;
    size_t out_sH = out_W;
    size_t out_sW = 1;
  
    CUDA_KERNEL_LOOP_TYPE(index, nthreads, size_t) {
        const size_t w = index % out_W;
        const size_t h = (index / out_W) % out_H;
        const size_t n = index / (out_H * out_W);
        const size_t grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;
  
        // get the corresponding input x, y co-ordinates from grid
        scalar_t ix = grid[grid_offset];
        scalar_t iy = grid[grid_offset + grid_sCoor];
  
        ix = grid_sampler_compute_source_index(ix, inp_W, padding_mode, align_corners);
        iy = grid_sampler_compute_source_index(iy, inp_H, padding_mode, align_corners);
  
        if (interpolation_mode == Interpolation::Bilinear) {
            // get NE, NW, SE, SW pixel values from (x, y)
            size_t ix_nw = static_cast<size_t>(::floor(ix));
            size_t iy_nw = static_cast<size_t>(::floor(iy));
            size_t ix_ne = ix_nw + 1;
            size_t iy_ne = iy_nw;
            size_t ix_sw = ix_nw;
            size_t iy_sw = iy_nw + 1;
            size_t ix_se = ix_nw + 1;
            size_t iy_se = iy_nw + 1;
    
            // get surfaces to each neighbor:
            scalar_t nw = (ix_se - ix)    * (iy_se - iy);
            scalar_t ne = (ix    - ix_sw) * (iy_sw - iy);
            scalar_t sw = (ix_ne - ix)    * (iy    - iy_ne);
            scalar_t se = (ix    - ix_nw) * (iy    - iy_nw);
    
            // calculate bilinear weighted pixel value and set output pixel
            auto inp_ptr_NC = input + n * inp_sN;
            auto out_ptr_NCHW = output + n * out_sN + h * out_sH + w * out_sW;
            for (size_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
                *out_ptr_NCHW = static_cast<scalar_t>(0);
                if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
                    *out_ptr_NCHW += inp_ptr_NC[iy_nw * inp_sH + ix_nw * inp_sW] * nw;
                }
                if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
                    *out_ptr_NCHW += inp_ptr_NC[iy_ne * inp_sH + ix_ne * inp_sW] * ne;
                }
                if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
                    *out_ptr_NCHW += inp_ptr_NC[iy_sw * inp_sH + ix_sw * inp_sW] * sw;
                }
                if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
                    *out_ptr_NCHW += inp_ptr_NC[iy_se * inp_sH + ix_se * inp_sW] * se;
                }
            }
        } 
        else if (interpolation_mode == Interpolation::Nearest) {
            size_t ix_nearest = static_cast<size_t>(::round(ix));
            size_t iy_nearest = static_cast<size_t>(::round(iy));
    
            // assign nearest neighor pixel value to output pixel
            auto inp_ptr_NC = input + n * inp_sN;
            auto out_ptr_NCHW = output + n * out_sN + h * out_sH + w * out_sW;
            for (size_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
                if (within_bounds_2d(iy_nearest, ix_nearest, inp_H, inp_W)) {
                    *out_ptr_NCHW = inp_ptr_NC[iy_nearest * inp_sH + ix_nearest * inp_sW];
                } else {
                    *out_ptr_NCHW = static_cast<scalar_t>(0);
                }
            }
        }
    }
}

int GridSamplerPlugin::enqueue(const nvinfer1::PluginTensorDesc* inputDesc, const nvinfer1::PluginTensorDesc* outputDesc,
	const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream)
{
    const int64_t count = inputDesc[0].dims.d[0] * inputDesc[0].dims.d[2] * inputDesc[0].dims.d[3];

    if (inputDesc[0].type == nvinfer1::DataType::kFLOAT)
    {
        grid_sampler_kernel<<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count,
            reinterpret_cast<const float*>(inputs[0]), inputDesc[0].dims.d[1], inputDesc[0].dims.d[2], inputDesc[0].dims.d[3],
            reinterpret_cast<const float*>(inputs[1]),
            reinterpret_cast<float*>(outputs[0]), outputDesc[0].dims.d[2], outputDesc[0].dims.d[3],
            m_interpolation_mode, m_padding_mode, m_align_corners);
    }
    // else if (inputDesc[0].type == nvinfer1::DataType::kHALF)
    // {
    //     grid_sampler_kernel<<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count,
    //         reinterpret_cast<const __half*>(inputs[0]), inputDesc[0].dims.d[1], inputDesc[0].dims.d[2], inputDesc[0].dims.d[3],
    //         reinterpret_cast<const __half*>(inputs[1]),
    //         reinterpret_cast<__half*>(outputs[0]), outputDesc[0].dims.d[2], outputDesc[0].dims.d[3],
    //         m_interpolation_mode, m_padding_mode, m_align_corners);
    // }

    return hipGetLastError();
}
